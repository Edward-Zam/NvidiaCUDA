#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"

__global__
void getMax(const float* const d_logLuminance, float* const d_bout, const size_t numels )
{
	extern __shared__ float sdata[];
	int id	= blockDim.x * blockIdx.x * threadIdx.x;
	int tid	= threadIdx.x; 
	sdata[tid] = d_logLuminance[id];
	__syncthreads();
	
	for ( int  i = 0; i <= numels - 1; i++)
	{
		id = threadId.x;
		stride = 1;
		offset = 2;
		if((threadIdx.x + 1)%offset == 0)
		{
			sdata[id]	= sdata[tid] > sdata[tid-stride] ? sdata[tid]:sdata[tid-stride];
			//sdata[tid] = max(sdata[tid], sdata[tid-stride];
		}
		
		__syncthreads();
		stride = stride*2;
		offset = offset^2;
	}
	
	if (tid == numels)
	{
		d_bout[blockIdx.x] = sdata[numels];
	}
}

__global__
void getMin(const float* const d_logLuminance, float* const d_bout, const size_t numels )
{
	extern __shared__ float sdata[];
	int id	= blockDim.x * blockIdx.x * threadIdx.x;
	int tid	= threadIdx.x; 
	sdata[tid] = d_logLuminance[id];
	__syncthreads();
	
	for ( int  i = 0; i <= numels - 1; i++)
	{
		id = threadId.x;
		stride = 1;
		offset = 2;
		if((threadIdx.x + 1)%offset == 0)
		{
			sdata[id]	= sdata[tid] > sdata[tid-stride] ? sdata[tid-stride]:sdata[tid];
			//sdata[tid] = min(sdata[tid], sdata[tid-stride];
		}
		
		__syncthreads();
		stride = stride*2;
		offset = offset^2;
	}
	
	if (tid == numels)
	{
		d_bout[blockIdx.x] = sdata[numels];
	}
}

__global__
void getHist(int *d_bins, const int *d_in, const float minLum, const float lumRange, const int BIN_COUNT)
{
	// accumulate using atomics to avoid race conditions
	int myId	= threadIdx.x + blockDim.x * blockIdx.x;
	int myItem	= d_in[myId];
	int myBin	= (myItem - minLum) / lumRange * BIN_COUNT;
	atomicAdd(&(d_bins[myBin]),1);
}

__global__
void HillisSteeleScan()
{
	// Copy bins to local memory
	extern __shared__ unsigned int t_in[];
	
	int id	= blockIdx.x * blockDim.x + threadIdx.x;
	int tid	= threadIdx.x;
	
	t_in[tid] = d_in[id];
	__syncthreads();
	
	for(unsigned int j = 1; j < BIN_COUNT; j<<1)
	{
		if(tid+j<blockDim.x)
		{
			atomicAdd(t_in[tid+j], t_in[tid]);
			__syncthreads();
		}
		d_out[id] = tid>0?t_in[tid-1]:0;
	}
	
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
	   
	float *d_temp;
	float *d_luminanceOut;
	float lumRange;
	
	unsigned int *d_bins;
	
	// Calculate number of elements in the array;
	const size_t numels = numCols * numRows;
	// Instatiate threads to max allowed per block for this GPU
	int threads = 1024;
	// Calculate number of blocks
	int blocks = numels/threads;

	// Allocate device memory
	checkCudaErrors(hipMalloc(&d_temp, sizeof(float)*blocks))
	checkCudaErrors(hipMalloc(&d_logLuminance, sizeof(float)*1));
	// Set memory to zero
	//checkCudaErrors(hipMemset())
	
	// Find the minimum value in the input logLuminance channel
	getMin<<<blocks, threads, threads * sizeof(float)>>>(d_logLuminance, d_temp, numels);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	threads = blocks;
	blocks = 1;
	getMin<<<blocks, threads,threads * sizeof(float)>>>(d_temp,d_luminanceOut,threads);
	hipDeviceSynchronize();checkCudaErrors(hipGetLastError());
	// Copy minimum luminance value to min_logLum in host memory
	checkCudaErrors(hipMemcpy(&min_logLum, d_luminanceOut, sizeof(float), hipMemcpyDeviceToHost));
	
	// Reset thread and block values
	threads = 1024;
	blocks = numels/threads;
	// Find the maximum value in the input logLuminance channel
	getMax<<<blocks, threads, threads * sizeof(float)>>>(d_logLuminance, d_temp, numels);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	threads = blocks;
	blocks = 1;
	getMax<<<blocks, threads, threads * sizeof(float)>>>(d_temp, d_luminanceOut, threads);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	// Copy maximum luminance value to max_logLum in host memory
	checkCudaErrors(hipMemcpy(&max_logLum, d_luminanceOut, sizeof(float), hipMemcpyDeviceToHost));
	
	// Calculate the luminance range
	lumRange = max_logLum - min_logLum;
	
	// Genereate a histogram of all values in the logLuminance channel
	threads = 1024;
	blocks = numels/threads;
	getHist<<<blocks, threads>>>(d_bins, d_logLuminance, min_logLum, lumRange, numBins);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	
	// Scan algorithm 
	// We are to perform an exclusive scan on the histogram; the histogram has numBins elements 
	// and therefore can be launched as one block with numBins threads. 
	threads = numBins;
	blocks = 1;
	
	
	
	
}
