#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

const int N = 1024;	// matrix is N*N
const int K = 16;	// each block will be 32 * 32 blocks

// Utility functions: compare, print, and fill matrices
#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error at: %s : %d\n", file,line);
    fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);;
    exit(1);
  }
}

int compare_matrices(float *gpu, float *ref)
{
	int result = 0;

	for(int j=0; j < N; j++)
    	for(int i=0; i < N; i++)
    		if (ref[i + j*N] != gpu[i + j*N])
    		{
    			// printf("reference(%d,%d) = %f but test(%d,%d) = %f\n",
    			// i,j,ref[i+j*N],i,j,test[i+j*N]);
    			result = 1;
    		}
    return result;
}

void print_matrix(float *mat)
{
	for(int j=0; j < N; j++) 
	{
		for(int i=0; i < N; i++) { printf("%4.4g ", mat[i + j*N]); }
		printf("\n");
	}	
}

// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat)
{
	for(int j=0; j < N * N; j++)
		mat[j] = (float) j;
}

void transpose_CPU(float in [], float out[])
{
	for(int j=0;j<N;j++)
		for(int i=0;i<N;i++)
			out[j + i*N] = in[i + j*N]; //out(j,i) = in(i,j)
}

__global__ void 
transpose_serial(float in [], float out[])
{
	for(int j=0;j<N;j++)
		for(int i=0;i<N;i++)
			out[j + i*N] = in[i + j*N]; //out(j,i) = in(i,j)
}

__global__ void
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;
	
	for(int j=0; j<N;j++)
		out[i + i*N] = in[i + j*N];
}

__global__ void
transpose_parallel_per_element(float in[], float out[])
{
	int i	= blockIdx.x * blockDim.x + threadIdx.x;
	int j	= blockIdx.y * blockDim.y + threadIdx.y;
	
	out[j + i*N] = in[i + j*N];
}

// to be launched with one thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elmts
__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{	
	// corner in and corner out are transposed
	int corner_i_in		= blockIdx.x * K, corner_j_in  = blockIdx.y * K;
	int corner_i_out	= blockIdx.y * K, corner_j_out = blockIdx.x * K;
	
	int x = threadIdx.x, y = threadIdx.y;
	
	__shared__ float tile[K][K];
	
	// coalesced read from global memory, transposed write to shared mem
	tile[y][x] = in[(corner_i_in + x)+ (corner_j_in + y) * N];
	__syncthreads();
	
	// read from shared memory, coalesced write
	out[(corner_i_out + x) + (corner_j_out + y) * N] = tile[x][y];
	
}

int main(int argc, char **argv)
{
	int numbytes = N*N*sizeof(float);
	
	float *in	= (float *) malloc(numbytes);
	float *out	= (float *) malloc(numbytes);
	float *gold	= (float *) malloc(numbytes);
	
	fill_matrix(in);
	transpose_CPU(in, gold);
	
	float *d_in, *d_out;
	
	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);
	
	GpuTimer timer;
	timer.Start();
	transpose_serial<<<1,1>>>(d_in, d_out); //takes about 0.084768 ms for N = 8; 
	// transpose_serial: 466.429 ms for N = 1024;
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_serial: %g ms. \nVerifying transpose...%s\n,", 
		timer.Elapsed(),
		compare_matrices(out,gold) ? "Failed": "Success"); 
		
	timer.Start();	
	transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
	// transpose_parallel_per_row: 4.76848 ms
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms. \nVerifying transpose...%s\n,", 
		timer.Elapsed(),
		compare_matrices(out,gold) ? "Failed": "Success"); 
		
	dim3 blocks(N/K, N/K);
	dim3 threads(K,K);
	timer.Start();
	transpose_parallel_per_element<<<blocks, threads>>>(d_in, d_out);
	// transpose_parallel_per_element: 0.341504 ms for N=1024
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element: %g ms. \nVerifying tranpose...%s\n,",
		timer.Elapsed(),
		compare_matrices(out,gold) ? "Failed": "Success");
		
	timer.Start();
	transpose_parallel_per_element_tiled<<<blocks,threads>>>(d_in,d_out);
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element_tiled: %g ms. \nVerifying transpose...%s\n,", 
	timer.Elapsed(),
	compare_matrices(out,gold)?"Failed":"Success");
	
//	printf("input matrix:\n"); print_matrix(in);
//	printf("reference or 'gold' transposed matrix:\n"); print_matrix(gold);
	
	
}
