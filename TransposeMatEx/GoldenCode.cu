#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

const int N = 1024;	// matrix is N*N
const int K = 32;	// each block will be 32 * 32 blocks

void transpose_CPU(float in [], float out[])
{
	for(int j=0;j<N;j++)
		for(int i=0;i<N;i++)
			out[j + i*N] = in[i + j*N]; //out(j,i) = in(i,j)
}

__global__ void 
transpose_serial(float in [], float out[])
{
	for(int j=0;j<N;j++)
		for(int i=0;i<N;i++)
			out[j + i*N] = in[i + j*N]; //out(j,i) = in(i,j)
}

__global__ void
transpose_parallel_per_row(float in[], float out[])
{
	int i = threadIdx.x;
	
	for(int j=0; j<N;j++)
		out[i + i*N] = in[i + j*N];
}

__global__ void
transpose_parallel_per_element(float in[], float out[])
{
	int i	= blockIdx.x * blockDim.x + threadIdx.x;
	int j	= blockIdx.y * blockDim.y + threadIdx.y;
	
	out[j + i*N] = in[i + j*N];
}

int main(int argc, char **argv)
{
	int numbytes = N*N*sizeof(float);
	
	float *in	= (float *) malloc(numbytes);
	float *out	= (float *) malloc(numbytes);
	float *gold	= (float *) malloc(numbytes);
	
	fill_matrix(in);
	transpose_CPU(in, gold);
	
	float *d_in, *d_out;
	
	hipMalloc(&d_in, numbytes);
	hipMalloc(&d_out, numbytes);
	hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);
	
	GpuTimer timer;
	timer.Start();
	transpose_serial<<<1,1>>>(d_in, d_out); //takes about 0.084768 ms for N = 8; 
	// transpose_serial: 466.429 ms for N = 1024;
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_serial: %g ms. \nVerifying transpose...%s\n,", 
		timer.Elapsed(),
		compare_matrices(out,gold) ? "Failed": "Success"); 
		
	timer.Start();	
	transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
	// transpose_parallel_per_row: 4.76848 ms
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_row: %g ms. \nVerifying transpose...%s\n,", 
		timer.Elapsed(),
		compare_matrices(out,gold) ? "Failed": "Success"); 
		
	dim3 blocks(N/K, N/K);
	dim3 threads(K,K);
	timer.Start();
	transpose_parallel_per_element<<<blocks, threads>>>(d_in, d_out);
	// transpose_parallel_per_element: 0.341504 ms for N=1024
	timer.Stop();
	hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
	printf("transpose_parallel_per_element: %g ms. \nVerifying tranpose...%s\n,",
		timer.Elapsed(),
		compare_matrices(out,gold) ? "Failed": "Success");
	
//	printf("input matrix:\n"); print_matrix(in);
//	printf("reference or 'gold' transposed matrix:\n"); print_matrix(gold);
	
	
}
